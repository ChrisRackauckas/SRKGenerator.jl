
#include <hip/hip_runtime.h>
// filename: integration.cu
// Performs the inner integration loop
extern "C"
{
  __device__ int innerFunc(const float *coefs,const float i,const float isq2,const float isq3,const float isq4,const float isq5,const float isq6,const float isq7,const float isq8,const float j,const float jsq2,const float jsq3,const float jsq4,const float jsq5,const float jsq6,const float jsq7,const float jsq8)
  {
    return abs(coefs[0]*(jsq2) + coefs[1]*(jsq3) + coefs[2]*(jsq4) + coefs[3]*(jsq5) + coefs[4]*jsq6 + coefs[5]*jsq7 + coefs[6]*jsq8 + coefs[7]*(i) + coefs[8]*(i)*(jsq2) + coefs[9]*i*jsq3 + coefs[10]*(i)*(jsq4) + coefs[11]*i*jsq5 + coefs[12]*(i)*(jsq6) + coefs[13]*i*jsq7 + coefs[14]*(isq2) + coefs[15]*(isq2)*(jsq2) + coefs[16]*isq2*jsq3 + coefs[17]*(isq2)*(jsq4) + coefs[18]*isq2*jsq5 + coefs[19]*(isq2)*(jsq6) + coefs[20]*(isq3) + coefs[21]*(isq3)*(jsq2) + coefs[22]*isq3*jsq3 + coefs[23]*(isq3)*(jsq4) + coefs[24]*isq3*jsq5 + coefs[25]*(isq4) + coefs[26]*(isq4)*(jsq2) + coefs[27]*isq4*jsq3 + coefs[28]*(isq4)*(jsq4) + coefs[29]*(isq5) + coefs[30]*(isq5)*(jsq2) + coefs[31]*isq5*jsq3+ coefs[32]*(isq6) + coefs[33]*(isq6)*(jsq2) + coefs[34]*(isq7) + coefs[35]*(isq8))<1;
  }
    __global__ void integration(const float *coefs, const float *iArr, const float *jArr, const int sizei, const int sizej, const int equalDiv,const int startIdx, int *tmp)
    {
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        int globalIndex = index*equalDiv+startIdx;
        int loopInd;
        float i;
        float j;
        float isq2;
        float isq3;
        float isq4;
        float isq5;
        float isq6;
        float isq7;
        float isq8;
        float jsq2;
        float jsq3;
        float jsq4;
        float jsq5;
        float jsq6;
        float jsq7;
        float jsq8;
        int ans = 0;
        for(loopInd=0;loopInd<equalDiv;loopInd=loopInd+1){
          i = iArr[(globalIndex+loopInd)/sizej];
          j = jArr[(globalIndex+loopInd)%sizej];
          if(globalIndex+loopInd >= sizei*sizej){
            break;
          }
          if((globalIndex+loopInd)%sizej==0 || loopInd==0){
            isq2 = i*i;
            isq3 = i*isq2;
            isq4 = isq2*isq2;
            isq5 = i*isq4;
            isq6 = isq4*isq2;
            isq7 = i*isq6;
            isq8 = isq4*isq4;
          }
          jsq2 = j*j;
          jsq3 = j*jsq2;
          jsq4 = jsq2*jsq2;
          jsq5 = j*jsq4;
          jsq6 = jsq2*jsq4;
          jsq7 = j*jsq6;
          jsq8 = jsq4*jsq4;
          ans = ans + innerFunc(coefs,i,isq2,isq3,isq4,isq5,isq6,isq7,isq8,j,jsq2,jsq3,jsq4,jsq5,jsq6,jsq7,jsq8);
        }
        tmp[index] = ans;
    }
}
